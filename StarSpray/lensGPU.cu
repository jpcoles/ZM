
#include <hip/hip_runtime.h>
#if defined(HAVE_GPU) && defined(HAVE_NVIDIA_GPU)

#include <assert.h>
#include <math.h>
#include <string.h>
#include "lens.h"
#include <cutil.h>

#define BLOCK_SIZE 8

__global__ 
void lensd(int h,  int w,  unsigned char *image, 
           int sh, int sw, unsigned char *source,
           int cred, int *cmass, float thee);

__host__
void lens(int h, int w, unsigned char *image, int *mass,
          int sh, int sw, unsigned char *source,
          int cred, int *cmass)
  /* assumes these sizes:
     image[h][w], mass[h][w], source[sh][sw], cmass[h/cred][w/cred]
  */
{ 
    int size;
    unsigned char *imaged, *sourced;
    int *cmassd;


    int x,y;
    int z,cz;        /* 1D array indices */
    float thee=0;     /* thee means theta_E */
    float cw = w/cred;
    /* reduce mass density to coarse grid */
    float Mtotal=0;
    for (y=0; y<h; y++)
    {
        int cy = (float)y/cred;
        for (x=0; x<w; x++)
        { 
            int cx = (float)x/cred; 
            z  = w*y + x;
            cz = cw*cy + cx;
            float m = mass[z];
            cmass[cz] += m;
            Mtotal    += m;
        }
    }

    //fprintf(stderr, "ch=%f cw=%f\n", ch, cw);

    //thee = .35 * h*h / thee;    /* set theta_E to 1/3 the height */
    if (Mtotal != 0) 
        thee = .33 * h/Mtotal;    /* set theta_E to 1/3 the height */
    else
        thee = 0;

    size = sh * sw * 3 * sizeof(char);
    //fprintf(stderr, "size=%i\n", size);
    CUDA_SAFE_CALL(cudaMalloc((void **)&sourced, size));
    CUDA_SAFE_CALL(cudaMemcpy(sourced, source, size, cudaMemcpyHostToDevice));

    size = h/cred * w/cred * sizeof(int);
    //fprintf(stderr, "size=%i\n", size);
    CUDA_SAFE_CALL(cudaMalloc((void **)&cmassd, size));
    CUDA_SAFE_CALL(cudaMemcpy(cmassd, cmass, size, cudaMemcpyHostToDevice));

    size = h * w * 4 * sizeof(char);
    //fprintf(stderr, "size=%i\n", size);
    CUDA_SAFE_CALL(cudaMalloc((void **)&imaged, size));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(w/dimBlock.x, h/dimBlock.y);

    //fprintf(stderr, "dimGrid.x=%i dimGrid.y=%i\n", dimGrid.x, dimGrid.y);

    lensd<<<dimGrid, dimBlock>>>(h, w, imaged, sh, sw, sourced, cred, cmassd, thee);

    CUDA_SAFE_CALL(cudaMemcpy(image, imaged, size, cudaMemcpyDeviceToHost));

    cudaFree(imaged);
    cudaFree(sourced);
    cudaFree(cmassd);
}

__global__ 
void lensd(int h,  int w,  unsigned char *image, 
           int sh, int sw, unsigned char *source,
           int cred, int *cmass, float thee)
{
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (x > w || y > h) return;

    int sx = x + (sw - w)/2.0;  
    int sy = y + (sh - h)/2.0;  /* align centers */

    int cx, cy;
    float ch = h/cred;
    float cw = w/cred;
    int *cmptr = cmass;
    for (cy=0; cy<ch; cy++)
    {
        float dy = y - cy*cred - cred/2;
        float dy2 = dy*dy;
        for (cx=0; cx<cw; cx++)
        { 
            float dx = x - cx*cred;
            float fac = (*cmptr++) * thee/sqrtf((dy2 + dx*dx + 10));
            sx -= dx*fac; 
            sy -= dy*fac;
        }
    }

    int i_offs = 4 * (y * w + x);
    if (0 < sx && sx < sw-1 && 0 < sy && sy < sh-1)
    { 
        int sz = (int)(sw*sy + sx) * 3;
        image[i_offs+0] = source[sz+0];
        image[i_offs+1] = source[sz+1];
        image[i_offs+2] = source[sz+2];
        image[i_offs+3] = 255;
    }
    else
    {
        image[i_offs+0] = 0;
        image[i_offs+1] = 0;
        image[i_offs+2] = 0;
        image[i_offs+3] = 255;
    }

}

#endif
